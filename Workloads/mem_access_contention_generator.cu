#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/syscall.h>
#include <unistd.h>
#include <sys/types.h>
#include <nvToolsExt.h>
#include <string.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

	__global__
void vertorADDGPU(int n, float *x, float *y, float *z) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// Memory reads of 8MB and memory writes of 4MB in turn
	// because n = 0x1 << 20
	if(idx < n) {
		z[idx] = x[idx] + y[idx];
	}
}

int main(int argc, char *argv[]) {
	int n = 0x1 << 20;
	int thr_num;
	float *x, *y, *z, *d_x, *d_y, *d_z;
	unsigned int flags = hipHostMallocMapped;

	// Check the number of arguments
	if (argc != 2) {
		printf("please use with one argument\n");
		printf("The argument is thread num\n");
		return 1;
	}
	thr_num = atoi(argv[1]);

	// Dynamic memory allocation by using zero-copy memory mechanism
	checkCudaErrors(hipHostAlloc((void **)&x, n * sizeof(float), flags));
	checkCudaErrors(hipHostAlloc((void **)&y, n * sizeof(float), flags));
	checkCudaErrors(hipHostAlloc((void **)&z, n * sizeof(float), flags));

	// Initialize the two input arrays
	for (int i = 0; i < n; i++) {
        x[i] = rand() / (float) RAND_MAX;
        y[i] = rand() / (float) RAND_MAX;
	}

	checkCudaErrors(hipHostGetDevicePointer((void **)&d_x, (void *)x, 0));
	checkCudaErrors(hipHostGetDevicePointer((void **)&d_y, (void *)y, 0));
	checkCudaErrors(hipHostGetDevicePointer((void **)&d_z, (void *)z, 0));

	// Set the number of threads
	dim3 block(thr_num);
	dim3 grid((unsigned int)ceil(n / (float)block.x));
	while (1) {
		vertorADDGPU<<<grid, block>>>(n, d_x, d_y, d_z);
	}

	checkCudaErrors(hipDeviceSynchronize());

	hipFree(d_x);
	hipFree(d_y);
	return 1;
}
